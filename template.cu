#include "hip/hip_runtime.h"
#include "CRtree.h"

__global__ void OddEvenSortPhaseKernel(float* d_arrayIn, int n, bool phase)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int subsetSize = 4;
	int indexBegin;

	if(phase) // FASE PAR
		indexBegin = id * (subsetSize*2);
	else // FASE IMPAR
		indexBegin = id * ((subsetSize*2)-subsetSize);

	if(indexBegin < n)
	{
		int indexEnd = indexBegin + (subsetSize*2);
		int i, temp;

		for(i = indexBegin; i < (indexEnd-1); i++)
		{
			if( i+1 < n &&
				d_arrayIn[i] > d_arrayIn[i+1] )
			{
				temp = d_arrayIn[i];
				d_arrayIn[i] = d_arrayIn[i+1];
				d_arrayIn[i+1] = temp;
			}
		}
	}
}

void OddEvenSortPhase(float* arrayIn, int n, bool phase)
{
	int size = n * sizeof(float);
	float *d_arrayIn;

	hipMalloc((void **) &d_arrayIn, size);
	hipMemcpy(d_arrayIn, arrayIn, size, hipMemcpyHostToDevice);

	dim3 DimGrid(ceil(n/256.0), 1, 1);
	dim3 DimBlock(256, 1, 1);
	OddEvenSortPhaseKernel<<< DimGrid, DimBlock >>>(d_arrayIn, n, phase);

	hipMemcpy(arrayIn, d_arrayIn, size, hipMemcpyDeviceToHost);

	hipFree(d_arrayIn);
}

int main()
{
	printf("Begin ==========\n");

	int numEntries = 4;
	int numFeatures = 64;
	CRtree rtree(numEntries, numFeatures);

	// INSERTANDO DATOS ==================================================
	float* bufferIn;
	bufferIn = (float*) malloc(numFeatures * sizeof(float));

	for(int i = 0; i < numFeatures; i++)
		bufferIn[i] = 0.0;

	//printf("root: %d\n", rtree.m_root->m_typeNode);

	rtree.InsertData(bufferIn);
	rtree.InsertData(bufferIn);
	rtree.InsertData(bufferIn);
	rtree.InsertData(bufferIn);
	rtree.InsertData(bufferIn);

	//printf("root: %d\n", rtree.m_root->m_typeNode);

	printf("End ============\n");

	return 0;
}
